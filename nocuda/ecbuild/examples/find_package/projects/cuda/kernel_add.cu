#include "hip/hip_runtime.h"
#include "kernel_add.h"

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__ void kernel_add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}
